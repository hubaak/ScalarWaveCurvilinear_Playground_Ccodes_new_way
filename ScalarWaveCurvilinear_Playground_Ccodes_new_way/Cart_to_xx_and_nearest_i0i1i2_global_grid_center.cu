#include "hip/hip_runtime.h"
#include "././NRPy_basic_defines.h"
/*
 * Given Cartesian point (x,y,z), this function outputs the corresponding
 *   (xx0,xx1,xx2) and the "closest" (i0,i1,i2) for the given grid
 */
 __host__  void Cart_to_xx_and_nearest_i0i1i2_global_grid_center(const paramstruct *restrict params, const REAL xCart[3], REAL xx[3], int Cart_to_i0i1i2[3]) {
#include "./set_Cparameters.h"

    const REAL Cartx = xCart[0];
    const REAL Carty = xCart[1];
    const REAL Cartz = xCart[2];

  /*
   *  Original SymPy expressions:
   *  "[xx[0] = SINHW*asinh(sqrt(Cartx**2 + Carty**2 + Cartz**2)*sinh(1/SINHW)/AMPL),
   *    xx[1] = acos(Cartz/sqrt(Cartx**2 + Carty**2 + Cartz**2)),
   *    xx[2] = atan2(Carty, Cartx)]"
   */
  const double tmp_0 = sqrt(((Cartx)*(Cartx)) + ((Carty)*(Carty)) + ((Cartz)*(Cartz)));
  xx[0] = SINHW*asinh(tmp_0*sinh((1.0/(SINHW)))/AMPL);
  xx[1] = acos(Cartz/tmp_0);
  xx[2] = atan2(Carty, Cartx);

    // Then find the nearest index (i0,i1,i2) on underlying grid to (x,y,z)
    // Recall that:
    // xx[0][j] = xxmin[0] + ((REAL)(j-NGHOSTS) + (1.0/2.0))*params->dxx0; // Cell-centered grid.
    //   --> j = (int) ( (xx[0][j] - xxmin[0]) / params->dxx0 + (1.0/2.0) + NGHOSTS )
    Cart_to_i0i1i2[0] = (int)( ( xx[0] - (0) ) / params->dxx0 + (1.0/2.0) + NGHOSTS - 0.5 ); // Account for (int) typecast rounding down
    Cart_to_i0i1i2[1] = (int)( ( xx[1] - (0) ) / params->dxx1 + (1.0/2.0) + NGHOSTS - 0.5 ); // Account for (int) typecast rounding down
    Cart_to_i0i1i2[2] = (int)( ( xx[2] - (-M_PI) ) / params->dxx2 + (1.0/2.0) + NGHOSTS - 0.5 ); // Account for (int) typecast rounding down
}
