#include "hip/hip_runtime.h"
#include "./NRPy_basic_defines.h"
#include "./NRPy_function_prototypes.h"
/*
 * Compute 1st derivative finite-difference derivative with arbitrary upwind
 */
__device__ static inline REAL FD1_arbitrary_upwind_x0_dirn(const paramstruct *restrict params, const REAL *restrict gf,
    const int i0,const int i1,const int i2, const int offset) {
#include "./set_Cparameters.h"

  switch(offset) {
  case 0:
    return (+1.0/12.0*gf[IDX3S(i0-2,i1,i2)]
            -2.0/3.0*gf[IDX3S(i0-1,i1,i2)]
            +2.0/3.0*gf[IDX3S(i0+1,i1,i2)]
            -1.0/12.0*gf[IDX3S(i0+2,i1,i2)]) * invdx0;
  case 1:
    return (-1.0/4.0*gf[IDX3S(i0-1,i1,i2)]
            -5.0/6.0*gf[IDX3S(i0,i1,i2)]
            +3.0/2.0*gf[IDX3S(i0+1,i1,i2)]
            -1.0/2.0*gf[IDX3S(i0+2,i1,i2)]
            +1.0/12.0*gf[IDX3S(i0+3,i1,i2)]) * invdx0;
  case -1:
    return (-1.0/12.0*gf[IDX3S(i0-3,i1,i2)]
            +1.0/2.0*gf[IDX3S(i0-2,i1,i2)]
            -3.0/2.0*gf[IDX3S(i0-1,i1,i2)]
            +5.0/6.0*gf[IDX3S(i0,i1,i2)]
            +1.0/4.0*gf[IDX3S(i0+1,i1,i2)]) * invdx0;
  case 2:
    return (-25.0/12.0*gf[IDX3S(i0,i1,i2)]
            +4*gf[IDX3S(i0+1,i1,i2)]
            -3*gf[IDX3S(i0+2,i1,i2)]
            +4.0/3.0*gf[IDX3S(i0+3,i1,i2)]
            -1.0/4.0*gf[IDX3S(i0+4,i1,i2)]) * invdx0;
  case -2:
    return (+1.0/4.0*gf[IDX3S(i0-4,i1,i2)]
            -4.0/3.0*gf[IDX3S(i0-3,i1,i2)]
            +3*gf[IDX3S(i0-2,i1,i2)]
            -4*gf[IDX3S(i0-1,i1,i2)]
            +25.0/12.0*gf[IDX3S(i0,i1,i2)]) * invdx0;
  }
  return 0.0 / 0.0;  // poison output if offset computed incorrectly
}
/*
 * Compute r(xx0,xx1,xx2).
 */
__device__  static inline void r_and_partial_xi_r_derivs(const paramstruct *restrict params,const REAL xx0,const REAL xx1,const REAL xx2,
                                  REAL *r, REAL *rinv, REAL *partial_x0_r,REAL *partial_x1_r,REAL *partial_x2_r) {
#include "./set_Cparameters.h"

  const double tmp_0 = (1.0/(SINHW));
  const double tmp_1 = exp(tmp_0) - exp(-tmp_0);
  const double tmp_3 = exp(tmp_0*xx0);
  const double tmp_4 = exp(-tmp_0*xx0);
  const double tmp_5 = tmp_3 - tmp_4;
  const double tmp_6 = tmp_1/AMPL;
  *r = AMPL*tmp_5/tmp_1;
  *rinv = tmp_6/tmp_5;
  *partial_x0_r = tmp_6/(tmp_0*tmp_3 + tmp_0*tmp_4);
  *partial_x1_r = 0;
  *partial_x2_r = 0;
}
/*
 * Compute \partial_r f
 */
__device__ static inline REAL compute_partial_r_f(const paramstruct *restrict params, REAL *restrict xx[3], const REAL *restrict gfs,
                                       const int which_gf, const int dest_i0,const int dest_i1,const int dest_i2,
                                       const int FACEi0,const int FACEi1,const int FACEi2,
                                       const REAL partial_x0_r, const REAL partial_x1_r, const REAL partial_x2_r) {
#include "./set_Cparameters.h"

  ///////////////////////////////////////////////////////////

  // FD1_stencil_radius = BC_FDORDER/2 = 2
  const int FD1_stencil_radius = 2;

  const int ntot = Nxx_plus_2NGHOSTS0*Nxx_plus_2NGHOSTS1*Nxx_plus_2NGHOSTS2;

  ///////////////////////////////////////////////////////////
  // Next we'll compute partial_xi f, using a maximally-centered stencil.
  //   The {i0,i1,i2}_offset parameters set the offset of the maximally-centered
  //   stencil, such that an offset=0 implies a centered stencil.

  // CHECK: Nxx_plus_2NGHOSTS0=10; FD1_stencil_radius=2. Then Nxx_plus_2NGHOSTS0-FD1_stencil_radius-1 = 7
  //  if dest_i0 = 9, we get i0_offset=7-9=-2, so the (4th order) deriv
  //  stencil is: -4,-3,-2,-1,0

  // CHECK: if FD1_stencil_radius=2 and dest_i0 = 1, we get i0_offset = FD1_stencil_radius-dest_i0 = 1,
  //  so the (4th order) deriv stencil is: -1,0,1,2,3

  // CHECK: if FD1_stencil_radius=2 and dest_i0 = 0, we get i0_offset = FD1_stencil_radius-1 = 2,
  //  so the (4th order) deriv stencil is: 0,1,2,3,4
  int i0_offset = FACEi0;  // up/downwind on the faces. This offset should never go out of bounds.
  if(dest_i0 < FD1_stencil_radius) i0_offset = FD1_stencil_radius-dest_i0;
  else if(dest_i0 > (Nxx_plus_2NGHOSTS0-FD1_stencil_radius-1)) i0_offset = (Nxx_plus_2NGHOSTS0-FD1_stencil_radius-1) - dest_i0;
  const REAL partial_x0_f=FD1_arbitrary_upwind_x0_dirn(params,&gfs[which_gf*ntot],dest_i0,dest_i1,dest_i2,i0_offset);

  const REAL partial_x1_f=0.0;
  const REAL partial_x2_f=0.0;
  return partial_x0_r*partial_x0_f + partial_x1_r*partial_x1_f + partial_x2_r*partial_x2_f;
}

/*
 * *** Apply radiation BCs to all outer boundaries. ***
 */
__device__ static inline void radiation_bcs_curvilinear(const paramstruct *restrict params, const bc_struct *restrict bcstruct,REAL *restrict xx[3],
                           const REAL *restrict gfs, REAL *restrict gfs_rhss,
                           const int which_gf, const int dest_i0,const int dest_i1,const int dest_i2,
                           const int FACEi0,const int FACEi1,const int FACEi2) {
#include "./set_Cparameters.h"
static const REAL gridfunctions_f_infinity_device[NUM_EVOL_GFS] = { 0.0, 0.0 };
static const REAL gridfunctions_wavespeed_device[NUM_EVOL_GFS] = { 1.0, 1.0 };


  // Nearest "interior" neighbor of this gridpoint, based on current face
  const int dest_i0_int=dest_i0+1*FACEi0, dest_i1_int=dest_i1+1*FACEi1, dest_i2_int=dest_i2+1*FACEi2;
  REAL r,rinv, partial_x0_r,partial_x1_r,partial_x2_r;
  REAL r_int,r_intinv, partial_x0_r_int,partial_x1_r_int,partial_x2_r_int;
  r_and_partial_xi_r_derivs(params,xx[0][dest_i0],    xx[1][dest_i1],    xx[2][dest_i2],    &r,    &rinv,    &partial_x0_r,    &partial_x1_r,    &partial_x2_r);
  r_and_partial_xi_r_derivs(params,xx[0][dest_i0_int],xx[1][dest_i1_int],xx[2][dest_i2_int],&r_int,&r_intinv,&partial_x0_r_int,&partial_x1_r_int,&partial_x2_r_int);
  const REAL partial_r_f     = compute_partial_r_f(params,xx,gfs, which_gf,dest_i0,    dest_i1,    dest_i2,
                                                   FACEi0,FACEi1,FACEi2,
                                                   partial_x0_r,    partial_x1_r,    partial_x2_r);
  const REAL partial_r_f_int = compute_partial_r_f(params,xx,gfs, which_gf,dest_i0_int,dest_i1_int,dest_i2_int,
                                                   FACEi0,FACEi1,FACEi2,
                                                   partial_x0_r_int,partial_x1_r_int,partial_x2_r_int);

  const int idx3 = IDX3S(dest_i0,dest_i1,dest_i2);
  const int idx3_int = IDX3S(dest_i0_int,dest_i1_int,dest_i2_int);

  const REAL partial_t_f_int = gfs_rhss[IDX4ptS(which_gf, idx3_int)];

  const REAL c = gridfunctions_wavespeed_device[which_gf];
  const REAL f_infinity = gridfunctions_f_infinity_device[which_gf];
  const REAL f = gfs[IDX4ptS(which_gf, idx3)];
  const REAL f_int = gfs[IDX4ptS(which_gf, idx3_int)];
  const REAL partial_t_f_int_outgoing_wave = -c * (partial_r_f_int + (f_int - f_infinity) * r_intinv);

  const REAL k = r_int*r_int*r_int * (partial_t_f_int - partial_t_f_int_outgoing_wave);

  const REAL partial_t_f_outgoing_wave = -c * (partial_r_f + (f - f_infinity) * rinv);

  gfs_rhss[IDX4ptS(which_gf, idx3)] = partial_t_f_outgoing_wave + k * rinv*rinv*rinv;
}

/*
 * Curvilinear boundary condition driver routine: Apply BCs to all six
 *   boundary faces of the 3D numerical domain, filling in the
 *   innermost ghost zone layer first, and moving outward.
 */
__device__  void apply_bcs_curvilinear_radiation(const paramstruct *restrict params, const bc_struct *restrict bcstruct,
                           const int NUM_GFS, const int8_t *restrict gfs_parity, REAL *restrict xx[3],
                           REAL *restrict gfs, REAL *restrict gfs_rhss) {
#include "./set_Cparameters.h"
#include "./set_cudaparameters.h"
  for(int which_gf=blockid; which_gf<NUM_GFS;which_gf+= Grid_size) {
    for(int which_gz = 0; which_gz < NGHOSTS; which_gz++) {

      // First apply OUTER boundary conditions,
      //   in case an INNER (parity) boundary point
      //   needs data at the outer boundary:
      // After updating each face, adjust imin[] and imax[]
      //   to reflect the newly-updated face extents.
      for(int pt=tid;pt<bcstruct->num_ob_gz_pts[which_gz];pt+=Block_size) {
        // *** Apply radiation BCs to all outer boundary points. ***
        radiation_bcs_curvilinear(params, bcstruct, xx, gfs, gfs_rhss,  which_gf,
                                  bcstruct->outer[which_gz][pt].outer_bc_dest_pt.i0,
                                  bcstruct->outer[which_gz][pt].outer_bc_dest_pt.i1,
                                  bcstruct->outer[which_gz][pt].outer_bc_dest_pt.i2,
                                  bcstruct->outer[which_gz][pt].FACEi0,
                                  bcstruct->outer[which_gz][pt].FACEi1,
                                  bcstruct->outer[which_gz][pt].FACEi2);
      }

      // Apply INNER (parity) boundary conditions:
      for(int pt=tid;pt<bcstruct->num_ib_gz_pts[which_gz];pt+=Block_size) {
        const int i0dest = bcstruct->inner[which_gz][pt].inner_bc_dest_pt.i0;
        const int i1dest = bcstruct->inner[which_gz][pt].inner_bc_dest_pt.i1;
        const int i2dest = bcstruct->inner[which_gz][pt].inner_bc_dest_pt.i2;
        const int i0src  = bcstruct->inner[which_gz][pt].inner_bc_src_pt.i0;
        const int i1src  = bcstruct->inner[which_gz][pt].inner_bc_src_pt.i1;
        const int i2src  = bcstruct->inner[which_gz][pt].inner_bc_src_pt.i2;
        gfs_rhss[IDX4S(which_gf,i0dest,i1dest,i2dest)] =
          bcstruct->inner[which_gz][pt].parity[gfs_parity[which_gf]] * gfs_rhss[IDX4S(which_gf, i0src,i1src,i2src)];
      } // END for(int pt=0;pt<num_ib_gz_pts[which_gz];pt++)
    } // END for(int which_gz = 0; which_gz < NGHOSTS; which_gz++)
  } // END for(int which_gf=0;which_gf<NUM_GFS;which_gf++)
}
