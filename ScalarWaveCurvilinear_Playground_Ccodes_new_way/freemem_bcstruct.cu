#include "./NRPy_basic_defines.h"
#include "./NRPy_function_prototypes.h"
/*
 * Free memory allocated within bcstruct
 */
 __host__ void freemem_bcstruct(const paramstruct *restrict params, const bc_struct *restrict bcstruct) {
#include "./set_Cparameters.h"

  for(int i=0;i<NGHOSTS;i++) { hipFree(bcstruct->outer[i]);  hipFree(bcstruct->inner[i]); }
  hipFree(bcstruct->outer);  hipFree(bcstruct->inner);
  hipFree(bcstruct->num_ob_gz_pts); hipFree(bcstruct->num_ib_gz_pts);
}
